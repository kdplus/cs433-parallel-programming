#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <assert.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}

const int TILE_DIM = 32;
const int NUM_REPS = 100;
const int SIDE = 8;
// Check errors and print GB/s
void postprocess(const float *ref, const float *res, int n, float ms)
{
	bool passed = true;
	for (int i = 0; i < n; i++)
		if (res[i] != ref[i]) {
			printf("%d %f %f\n", i, res[i], ref[i]);
			printf("%25s\n", "*** FAILED ***");
			passed = false;
			break;
		}
	if (passed)
		printf("%20.2f%20.2f\n", ms/NUM_REPS, 2 * n * sizeof(float) * 1e-6 * NUM_REPS / ms);
}

__global__ void matrixTranspose(float *_a, float *_b, const int cols, const int rows)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y; // row
	int j = blockIdx.x * blockDim.x + threadIdx.x; // col
	int index_in = i*cols + j; // (i,j) from matrix A
	int index_out = j*rows + i; // transposed index
	_b[index_out] = _a[index_in];
}

__global__ void matrixTransposeShared(const float *_a, float *_b, const int cols, const int rows)
{
	__shared__ float mat[TILE_DIM][TILE_DIM];
	int bx = blockIdx.x *blockDim.x;
	int by = blockIdx.y *blockDim.y;
	int i = by + threadIdx.y; int j = bx + threadIdx.x; //input
	int ti = bx + threadIdx.y; int tj = by + threadIdx.x;

	//output
	if (i < rows && j < cols)//i < ny && j < nx
		mat[threadIdx.x][threadIdx.y] = _a[i * cols + j];
	__syncthreads(); //Wait for all data to be copied
	if (tj < cols && ti < rows)
		_b[ti * rows + tj] = mat[threadIdx.y][threadIdx.x];
}

__global__ void matrixTransposeSharedwBC(const float *_a, float *_b, const int cols, const int rows)
{
	__shared__ float mat[TILE_DIM][TILE_DIM + 1];
	int bx = blockIdx.x *blockDim.x;
	int by = blockIdx.y *blockDim.y;
	int i = by + threadIdx.y; int j = bx + threadIdx.x; //input
	int ti = bx + threadIdx.y; int tj = by + threadIdx.x;

	//output
	if (i < rows && j < cols)//i < rows && j < cols
		mat[threadIdx.x][threadIdx.y] = _a[i * cols + j];
	__syncthreads(); //Wait for all data to be copied
	if (tj < cols && ti < rows)
		_b[ti * rows + tj] = mat[threadIdx.y][threadIdx.x];
}

__global__ void matrixTransposeUnrolled(const float *_a, float *_b, const int cols, const int rows)
{
	__shared__ float mat[TILE_DIM][TILE_DIM + 1];
	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
#pragma unroll
	for (int k = 0; k < TILE_DIM; k += SIDE) {
		if (x < rows && y + k < cols)
			mat[threadIdx.y + k][threadIdx.x] = _a[((y + k) * rows) + x];
	}

	__syncthreads();

	x = blockIdx.y * TILE_DIM + threadIdx.x;
	y = blockIdx.x * TILE_DIM + threadIdx.y;
#pragma unroll
	for (int k = 0; k < TILE_DIM; k += SIDE)
	{
		if (x < cols && y + k < rows)
			_b[(y + k) * cols + x] = mat[threadIdx.x][threadIdx.y + k];
	}
}

int main(int argc, char **argv){
	const int nx = 1024;
	const int ny = 1024;
	const int mem_size = nx*ny*sizeof(float);

	dim3 gridDim(nx / TILE_DIM, ny / TILE_DIM, 1);
	dim3 blockDim(TILE_DIM, TILE_DIM, 1);

	int devId = 0;
	if (argc > 1) devId = atoi(argv[1]);

	hipDeviceProp_t prop;
	checkCuda(hipGetDeviceProperties(&prop, devId));
	printf("\nDevice : %s\n", prop.name);
	printf("Matrix size: %d %d, Block size: %d %d\n",
		nx, ny, TILE_DIM, TILE_DIM);
	printf("gridDim: %d %d %d. blockDim: %d %d %d\n",
		gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);

	checkCuda(hipSetDevice(devId));

	float *h_idata = (float*)malloc(mem_size);
	float *h_tdata = (float*)malloc(mem_size);
	float *gold = (float*)malloc(mem_size);

	float *d_idata, *d_cdata, *d_tdata;
	checkCuda(hipMalloc(&d_idata, mem_size));
	checkCuda(hipMalloc(&d_tdata, mem_size));

	// host
	for (int j = 0; j < ny; j++)
		for (int i = 0; i < nx; i++)
			h_idata[j*nx + i] = j*nx + i;

	// correct result for error checking
	for (int j = 0; j < ny; j++)
		for (int i = 0; i < nx; i++)
			gold[j*nx + i] = h_idata[i*nx + j];

	// device
	checkCuda(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

	// events for timing
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));
	float ms;

	// ------------
	// time kernels
	// ------------
	printf("%25s%20s%25s\n", "Method","Time(ms)", "Bandwidth (GB/s)");
	
	// ----
	// matrixTranspose 
	// ----
	printf("%25s", "matrixTranspose");
	checkCuda(hipMemset(d_tdata, 0, mem_size));
	//matrixTranspose << <gridDim, blockDim >> >(d_idata, d_tdata, nx, ny);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		matrixTranspose << <gridDim, blockDim >> >(d_idata, d_tdata, nx, ny);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
	postprocess(gold, h_tdata, nx * ny, ms);

	// ----
	// matrixTransposeShared 
	// ----
	printf("%25s", "matrixTransposeShared");
	checkCuda(hipMemset(d_tdata, 0, mem_size));
	//matrixTransposeShared << <gridDim, blockDim >> >(d_idata, d_tdata, nx, ny);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		matrixTransposeShared << <gridDim, blockDim >> >(d_idata, d_tdata, nx, ny);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
	postprocess(gold, h_tdata, nx * ny, ms);
	
	// ----
	// matrixTransposeSharedwBC 
	// ----
	printf("%25s", "matrixTransposeSharedwBC");
	checkCuda(hipMemset(d_tdata, 0, mem_size));
	matrixTransposeSharedwBC << <gridDim, blockDim >> >(d_idata, d_tdata, nx, ny);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		matrixTransposeSharedwBC << <gridDim, blockDim >> >(d_idata, d_tdata, nx, ny);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
	postprocess(gold, h_tdata, nx * ny, ms);
	
	// ----
	// matrixTransposeUnrolled 
	// ----
	dim3 blockDimUnroll(TILE_DIM, SIDE, 1);// !important
	printf("Matrix size: %d %d, Block size: %d %d\n",
	nx, ny, TILE_DIM, TILE_DIM);
	printf("gridDim: %d %d %d. blockDim: %d %d %d\n",
	gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
	printf("%25s", "matrixTransposeUnrolled");
	checkCuda(hipMemset(d_tdata, 0, mem_size));
	//matrixTransposeUnrolled << <gridDim, blockDim >> >(d_idata, d_tdata, nx, ny);
	checkCuda(hipEventRecord(startEvent, 0));
	for (int i = 0; i < NUM_REPS; i++)
		matrixTransposeUnrolled << <gridDim, blockDimUnroll >> >(d_idata, d_tdata, nx, ny);
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
	checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
	postprocess(gold, h_tdata, nx * ny, ms);
	
error_exit:
	// cleanup
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	checkCuda(hipFree(d_tdata));
	checkCuda(hipFree(d_idata));
	free(h_idata);
	free(h_tdata);
	free(gold);
}